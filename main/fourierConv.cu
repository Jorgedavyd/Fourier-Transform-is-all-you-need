#include "hip/hip_runtime.h"
#include <cuFFT.h>
#include <cuBLAS.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <torch/extension.h>

struct params {
    const unsigned int i,
}
__host__ __forceinline__ struct define_kernel_parameters (void) {
    
}

template <typename T>
__global__ void conv2d_fwd_kernel (
    hipdnnTensorDescriptor_t<T> input,
    hipdnnTensorDescriptor_t<T> weight,
    hipdnnTensorDescriptor_t<T> bias,

) {

};


torch::Tensor conv2d_fwd (torch::Tensor[] input, torch::Tensor[] weight, torch::Tensor[] bias) {
    // cudnn handler
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // from torch::Tensor to cudnn tensor
    hipdnnTensorDescriptor_t in;
    hipdnnTensorDescriptor_t w;
    hipdnnTensorDescriptor_t b;
    hipdnnTensorDescriptor_t out;

    hipdnnGetTensor4dDescriptor(&in);
    hipdnnGetTensor4dDescriptor(&w);
    hipdnnGetTensor4dDescriptor(&b);
    hipdnnGetTensor4dDescriptor(&out);
    
    //Define the kernel based on the input

    // Run the kernel
    conv2d_fwd_kernel<<<gridSize, blockSize>>><float32>(&input, &weight, &bias);
    
    hipdnnDestroyTensorDescriptor(in);
    hipdnnDestroyTensorDescriptor(w);
    hipdnnDestroyTensorDescriptor(b);

    hipdnnDestroy(cudnn);

    return torch::from_blob(out);
};

__global__ void conv2d_bwd_kernel (void) {
    
}

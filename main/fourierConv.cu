#include "hip/hip_runtime.h"
#include <cuFFT.h>
#include <cuBLAS.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <torch/extension.h>

struct params {
    const unsigned int 
}
__host__ __forceinline__ struct define_kernel_parameters (void) {
    
}

template <typename T>
__global__ void conv2d_fwd_kernel (void) {
    

};


torch::Tensor conv2d_fwd (torch::Tensor[] input, torch::Tensor[] weight, torch::Tensor[] bias) {
    // from torch::Tensor to cudnn tensor
    hipdnnTensorDescriptor_t in;
    hipdnnTensorDescriptor_t w;
    hipdnnTensorDescriptor_t b;

    in = hipdnnGetTensor4dDescriptor();
    w = hipdnnGetTensor4dDescriptor();
    b = hipdnnGetTensor4dDescriptor();
    
    //Define the kernel based on the input

    // Run the kernel
    conv2d_fwd_kernel<<<gridSize, blockSize>>><float32>(&input, &weight, &bias);
};

__global__ void conv2d_bwd_kernel (void) {
    
}
